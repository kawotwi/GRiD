#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <pybind11/stl.h>
#include "grid.cuh" // Path to the grid.cuh header
#include <hip/hip_runtime.h>

namespace py = pybind11;

// Helper: convert numpy array to raw ptr
template <typename T>
T* get_data_ptr(py::array_t<T, py::array::c_style> array) {
    return static_cast<T*>(array.request().ptr);
}

// Python wrapper class for GRiD
template <typename T>
class PyGRidData {
private:
    T gravity;
    dim3 dimms;
    grid::gridData<T>* grid_data;
    hipStream_t* streams;
    grid::robotModel<T>* d_robot_model;

public:
    PyGRidData(T g = static_cast<T>(9.81)) {
        gravity = g;
        dimms = dim3(grid::SUGGESTED_THREADS, 1, 1);
        streams = grid::init_grid<T>();
        d_robot_model = grid::init_robotModel<T>();
        grid_data = grid::init_gridData<T, 1>();
    }

    ~PyGRidData() {
        grid::close_grid<T>(streams, d_robot_model, grid_data);
    }

    void load_joint_info(py::array_t<T, py::array::c_style> q,
                          py::array_t<T, py::array::c_style> qd,
                          py::array_t<T, py::array::c_style> u) {
        if (q.size() != grid::NUM_JOINTS || qd.size() != grid::NUM_JOINTS || u.size() != grid::NUM_JOINTS) {
            throw std::runtime_error("Input arrays must be of size NUM_JOINTS");
        }
        
        T* q_ptr = get_data_ptr(q);
        T* qd_ptr = get_data_ptr(qd);
        T* u_ptr = get_data_ptr(u);
        
        for (int i = 0; i < grid::NUM_JOINTS; i++) {
            grid_data->h_q_qd_u[i] = q_ptr[i];
            grid_data->h_q_qd_u[i + grid::NUM_JOINTS] = qd_ptr[i];
            grid_data->h_q_qd_u[i + 2 * grid::NUM_JOINTS] = u_ptr[i];
        }
        
        gpuErrchk(hipMemcpy(grid_data->d_q_qd_u, grid_data->h_q_qd_u,
                             3 * grid::NUM_JOINTS * sizeof(T), hipMemcpyHostToDevice));
        gpuErrchk(hipDeviceSynchronize());
    }

    // Get end effector positions
    py::array_t<T> get_end_effector_positions() {
        grid::end_effector_positions<T, false>(grid_data, d_robot_model, 1, dim3(1, 1, 1), dimms, streams);
        
        // Create numpy array from our data
        auto result = py::array_t<T>(6 * grid::NUM_EES);
        py::buffer_info buf = result.request();
        T* ptr = static_cast<T*>(buf.ptr);
        
        // Copy data from h_eePos to numpy array
        std::memcpy(ptr, grid_data->h_eePos, 6 * grid::NUM_EES * sizeof(T));
        
        return result;
    }

    // Get end effector position gradients
    py::array_t<T> get_end_effector_position_gradients() {
        grid::end_effector_positions_gradient<T, false>(grid_data, d_robot_model, 1, dim3(1, 1, 1), dimms, streams);
        
        // Create numpy array with shape (6, NUM_EES*NUM_JOINTS)
        std::vector<size_t> shape = {6, static_cast<size_t>(grid::NUM_EES * grid::NUM_JOINTS)};
        std::vector<size_t> strides = {sizeof(T), 6 * sizeof(T)};
        
        auto result = py::array_t<T>(shape, strides);
        py::buffer_info buf = result.request();
        T* ptr = static_cast<T*>(buf.ptr);
        
        // Copy data from h_deePos to numpy array
        std::memcpy(ptr, grid_data->h_deePos, 6 * grid::NUM_EES * grid::NUM_JOINTS * sizeof(T));
        
        return result;
    }

    py::array_t<T> inverse_dynamics() {
        grid::inverse_dynamics<T, false, false>(
            grid_data, d_robot_model, gravity, 1,
            dim3(1, 1, 1), dimms, streams
        );
        
        auto result = py::array_t<T>(grid::NUM_JOINTS);
        py::buffer_info buf = result.request();
        T* ptr = static_cast<T*>(buf.ptr);
        std::memcpy(ptr, grid_data->h_c, grid::NUM_JOINTS * sizeof(T));
        return result;
    }

    // Compute mass matrix inverse
    py::array_t<T> minv() {
        grid::direct_minv<T, false>(grid_data, d_robot_model, 1, dim3(1, 1, 1), dimms, streams);
        
        std::vector<size_t> shape = {
            static_cast<size_t>(grid::NUM_JOINTS), 
            static_cast<size_t>(grid::NUM_JOINTS)
        };
        auto result = py::array_t<T>(shape);
        py::buffer_info buf = result.request();
        T* ptr = static_cast<T*>(buf.ptr);
        
        std::memcpy(ptr, grid_data->h_Minv, grid::NUM_JOINTS * grid::NUM_JOINTS * sizeof(T));
        
        return result;
    }

    // Compute forward dynamics 
    py::array_t<T> forward_dynamics() {
        grid::forward_dynamics<T>(grid_data, d_robot_model, gravity, 1, dim3(1, 1, 1), dimms, streams);
        
        auto result = py::array_t<T>(grid::NUM_JOINTS);
        py::buffer_info buf = result.request();
        T* ptr = static_cast<T*>(buf.ptr);
        
        std::memcpy(ptr, grid_data->h_qdd, grid::NUM_JOINTS * sizeof(T));
        
        return result;
    }

    // Compute inverse dynamics gradient
    py::array_t<T> inverse_dynamics_gradient() {
        grid::inverse_dynamics_gradient<T, true, false>(grid_data, d_robot_model, gravity, 1, dim3(1, 1, 1), dimms, streams);
        
        // Create array for both dc_dq and dc_dqd
        std::vector<size_t> shape = {
            2,  // Two matrices: dc_dq and dc_dqd
            static_cast<size_t>(grid::NUM_JOINTS), 
            static_cast<size_t>(grid::NUM_JOINTS)
        };
        auto result = py::array_t<T>(shape);
        py::buffer_info buf = result.request();
        T* ptr = static_cast<T*>(buf.ptr);
        
        // Copy dc_dq (first NUM_JOINTS*NUM_JOINTS elements)
        std::memcpy(ptr, grid_data->h_dc_du, grid::NUM_JOINTS * grid::NUM_JOINTS * sizeof(T));
        
        // Copy dc_dqd (next NUM_JOINTS*NUM_JOINTS elements)
        std::memcpy(
            ptr + grid::NUM_JOINTS * grid::NUM_JOINTS, 
            &grid_data->h_dc_du[grid::NUM_JOINTS * grid::NUM_JOINTS], 
            grid::NUM_JOINTS * grid::NUM_JOINTS * sizeof(T)
        );
        
        return result;
    }

    // Compute forward dynamics gradient
    py::array_t<T> forward_dynamics_gradient() {
        grid::forward_dynamics_gradient<T, false>(grid_data, d_robot_model, gravity, 1, dim3(1, 1, 1), dimms, streams);
        
        // Create array for both df_dq and df_dqd
        std::vector<size_t> shape = {
            2,  // Two matrices: df_dq and df_dqd
            static_cast<size_t>(grid::NUM_JOINTS), 
            static_cast<size_t>(grid::NUM_JOINTS)
        };
        auto result = py::array_t<T>(shape);
        py::buffer_info buf = result.request();
        T* ptr = static_cast<T*>(buf.ptr);
        
        // Copy df_dq (first NUM_JOINTS*NUM_JOINTS elements)
        std::memcpy(ptr, grid_data->h_df_du, grid::NUM_JOINTS * grid::NUM_JOINTS * sizeof(T));
        
        // Copy df_dqd (next NUM_JOINTS*NUM_JOINTS elements)
        std::memcpy(
            ptr + grid::NUM_JOINTS * grid::NUM_JOINTS, 
            &grid_data->h_df_du[grid::NUM_JOINTS * grid::NUM_JOINTS], 
            grid::NUM_JOINTS * grid::NUM_JOINTS * sizeof(T)
        );
        
        return result;
    }
};

// Pybind11 module
PYBIND11_MODULE(gridCuda, m) {
    m.doc() = "Python bindings for CUDA GRiD dynamics";
    
    py::class_<PyGRidData<float>>(m, "GRidDataFloat")
        .def(py::init<float>(), py::arg("gravity") = 9.81f)
        .def("load_joint_info", &PyGRidData<float>::load_joint_info,
             "Load joint positions, velocities, torques",
             py::arg("q"), py::arg("qd"), py::arg("u"))
        .def("get_end_effector_positions", &PyGRidData<float>::get_end_effector_positions,
            "Calculate end effector positions")
        .def("get_end_effector_position_gradients", &PyGRidData<float>::get_end_effector_position_gradients,
            "Calculate end effector position gradients")
        .def("inverse_dynamics", &PyGRidData<float>::inverse_dynamics,
             "Compute inverse dynamics")
        .def("minv", &PyGRidData<float>::minv,
            "Calculate mass matrix inverse")
        .def("forward_dynamics", &PyGRidData<float>::forward_dynamics,
            "Calculate forward dynamics (joint accelerations)")
        .def("inverse_dynamics_gradient", &PyGRidData<float>::inverse_dynamics_gradient,
            "Calculate inverse dynamics gradient")
        .def("forward_dynamics_gradient", &PyGRidData<float>::forward_dynamics_gradient,
            "Calculate forward dynamics gradient");
    
    // double compatability was breaking code.
    // py::class_<PyGRidData<double>>(m, "GRidDataDouble")
    //     .def(py::init<double>(), py::arg("gravity") = 9.81)
    //     .def("load_joint_info", &PyGRidData<double>::load_joint_info,
    //          "Load joint positions, velocities, torques",
    //          py::arg("q"), py::arg("qd"), py::arg("u"))
    //     .def("inverse_dynamics", &PyGRidData<double>::inverse_dynamics,
    //          "Compute inverse dynamics");
    
    m.attr("NUM_JOINTS") = grid::NUM_JOINTS;
    m.attr("NUM_EES") = grid::NUM_EES;
}