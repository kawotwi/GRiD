#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <pybind11/stl.h>
#include "grid.cuh" // Path to the grid.cuh header
#include <hip/hip_runtime.h>

namespace py = pybind11;

// Helper: convert numpy array to raw ptr
template <typename T>
T* get_data_ptr(py::array_t<T, py::array::c_style> array) {
    return static_cast<T*>(array.request().ptr);
}

// Python wrapper class for GRiD
template <typename T>
class PyGRidData {
private:
    T gravity;
    dim3 dimms;
    grid::gridData<T>* grid_data;
    hipStream_t* streams;
    grid::robotModel<T>* d_robot_model;

public:
    PyGRidData(T g = static_cast<T>(9.81)) {
        gravity = g;
        dimms = dim3(grid::SUGGESTED_THREADS, 1, 1);
        streams = grid::init_grid<T>();
        d_robot_model = grid::init_robotModel<T>();
        grid_data = grid::init_gridData<T, 1>();
    }

    ~PyGRidData() {
        grid::close_grid<T>(streams, d_robot_model, grid_data);
    }

    void load_joint_info(py::array_t<T, py::array::c_style> q,
                          py::array_t<T, py::array::c_style> qd,
                          py::array_t<T, py::array::c_style> u) {
        if (q.size() != grid::NUM_JOINTS || qd.size() != grid::NUM_JOINTS || u.size() != grid::NUM_JOINTS) {
            throw std::runtime_error("Input arrays must be of size NUM_JOINTS");
        }
        
        T* q_ptr = get_data_ptr(q);
        T* qd_ptr = get_data_ptr(qd);
        T* u_ptr = get_data_ptr(u);
        
        for (int i = 0; i < grid::NUM_JOINTS; i++) {
            grid_data->h_q_qd_u[i] = q_ptr[i];
            grid_data->h_q_qd_u[i + grid::NUM_JOINTS] = qd_ptr[i];
            grid_data->h_q_qd_u[i + 2 * grid::NUM_JOINTS] = u_ptr[i];
        }
        
        gpuErrchk(hipMemcpy(grid_data->d_q_qd_u, grid_data->h_q_qd_u,
                             3 * grid::NUM_JOINTS * sizeof(T), hipMemcpyHostToDevice));
        gpuErrchk(hipDeviceSynchronize());
    }

    py::array_t<T> inverse_dynamics() {
        grid::inverse_dynamics<T, false, false>(
            grid_data, d_robot_model, gravity, 1,
            dim3(1, 1, 1), dimms, streams
        );
        
        auto result = py::array_t<T>(grid::NUM_JOINTS);
        py::buffer_info buf = result.request();
        T* ptr = static_cast<T*>(buf.ptr);
        std::memcpy(ptr, grid_data->h_c, grid::NUM_JOINTS * sizeof(T));
        return result;
    }
};

// Pybind11 module
PYBIND11_MODULE(gridCuda, m) {
    m.doc() = "Python bindings for CUDA GRiD dynamics";
    
    py::class_<PyGRidData<float>>(m, "GRidDataFloat")
        .def(py::init<float>(), py::arg("gravity") = 9.81f)
        .def("load_joint_info", &PyGRidData<float>::load_joint_info,
             "Load joint positions, velocities, torques",
             py::arg("q"), py::arg("qd"), py::arg("u"))
        .def("inverse_dynamics", &PyGRidData<float>::inverse_dynamics,
             "Compute inverse dynamics");
    
    // double compatability was breaking code.
    // py::class_<PyGRidData<double>>(m, "GRidDataDouble")
    //     .def(py::init<double>(), py::arg("gravity") = 9.81)
    //     .def("load_joint_info", &PyGRidData<double>::load_joint_info,
    //          "Load joint positions, velocities, torques",
    //          py::arg("q"), py::arg("qd"), py::arg("u"))
    //     .def("inverse_dynamics", &PyGRidData<double>::inverse_dynamics,
    //          "Compute inverse dynamics");
    
    m.attr("NUM_JOINTS") = grid::NUM_JOINTS;
}