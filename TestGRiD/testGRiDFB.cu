/***
nvcc -std=c++11 -o testGRiD.exe testGRiD.cu -gencode arch=compute_86,code=sm_86
***/

#include <iostream>
#include <algorithm>
#include <type_traits>
#include "../grid.cuh"

template <typename T>
__host__
void test(){
	T gravity = static_cast<T>(9.81);
	dim3 dimms(grid::SUGGESTED_THREADS,1,1);
	hipStream_t *streams = grid::init_grid<T>();
	grid::robotModel<T> *d_robotModel = grid::init_robotModel<T>();
	grid::gridData<T> *hd_data = grid::init_gridData<T,1>();

	// initialize values based on floating base or not
	T random_fb[90] = {0.300623, -1.427442, 0.047334, 0.2675568650090604, -0.6452699620984409, -0.001724583948895716, -0.7155676249036168, -1.226503, -0.619695, 0.973148, -0.750689, -0.253769, 0.493305, -0.695605, 0.425334, 0.340006, -0.178834, -0.013169, -2.349815, 0.405039, -2.266609, -0.424634, 1.034167, -0.270165, -0.18414, -1.111512, 0.659046, 0.183907, 0.944741, 0.579223, 0.497338, 0.870245, 1.098656, 1.553845, -1.160813, -2.30901, 0.501948, 1.172242, 0.451889, 0.883051, -0.662848, 0.038682, 0.814782, 1.139002, 0.2817, -1.699318, 0.72425, 0.503564, 0.78011, -0.424718, 0.736483, -1.500795, 0.636129, -0.351871, 0.029238, -1.177703, 0.329867, 0.684543, 0.223669, 1.556482, -0.477746, 2.010085, 0.26853, 1.4253, 1.747454, -0.317835, 0.336185, 0.752943, -0.506264, -2.587783, -0.356798, 0.154351, 2.536409, -0.547202, -1.094094, 0.600488, 0.473008, -0.033037, 0.095979, -1.173089, 0.04475, -1.920187, 0.656968, -0.625342, 0.762751, 1.943894, 1.846422, 0.207588, -0.233651, -0.57805};
	T random_nonfb[90] = {0.300623, -1.427442, 0.047334, -0.512040, -1.437442, 0.500384, -0.881586, -1.226503, -0.619695, 0.973148, -0.750689, -0.253769, 0.493305, -0.695605, 0.425334, 0.340006, -0.178834, -0.013169, -2.349815, 0.405039, -2.266609, -0.424634, 1.034167, -0.270165, -0.184140, -1.111512, 0.659046, 0.183907, 0.944741, 0.579223, 0.497338, 0.870245, 1.098656, 1.553845, -1.160813, -2.309010, 0.501948, 1.172242, 0.451889, 0.883051, -0.662848, 0.038682, 0.814782, 1.139002, 0.281700, -1.699318, 0.724250, 0.503564, 0.780110, -0.424718, 0.736483, -1.500795, 0.636129, -0.351871, 0.029238, -1.177703, 0.329867, 0.684543, 0.223669, 1.556482, -0.477746, 2.010085, 0.268530, 1.425300, 1.747454, -0.317835, 0.336185, 0.752943, -0.506264, -2.587783, -0.356798, 0.154351, 2.536409, -0.547202, -1.094094, 0.600488, 0.473008, -0.033037, 0.095979, -1.173089, 0.044750, -1.920187, 0.656968, -0.625342, 0.762751, 1.943894, 1.846422, 0.207588, -0.233651, -0.578050};
	T random[90];
	if (grid::NUM_JOINTS != grid::NUM_VEL) {
		for(int i = 0; i < 90; i++) random[i] = random_fb[i];
	}
	else {
		for(int i = 0; i < 90; i++) random[i] = random_nonfb[i];
	}
	if (grid::NUM_JOINTS > 30) {
		printf("GRiD does not support robots with > 30 dof"); 
		return;
	} 

	for (int i = 0; i < grid::NUM_JOINTS; i++) hd_data->h_q_qd_u[i] = random[i];
	for (int i = 0; i < grid::NUM_VEL; i++) hd_data->h_q_qd_u[grid::NUM_JOINTS+i] = random[i+grid::NUM_JOINTS];
	for (int i = 0; i < grid::NUM_VEL; i++) hd_data->h_q_qd_u[grid::NUM_JOINTS+grid::NUM_VEL+i] = static_cast<T>(0);


	gpuErrchk(hipMemcpy(hd_data->d_q_qd_u,hd_data->h_q_qd_u,3*grid::NUM_JOINTS*sizeof(T),hipMemcpyHostToDevice));
	gpuErrchk(hipDeviceSynchronize());

	// q,qd,u
	printMat<T,1,grid::NUM_JOINTS>(hd_data->h_q_qd_u,1);
	printf("\n");
	printMat<T,1,grid::NUM_VEL>(&hd_data->h_q_qd_u[grid::NUM_JOINTS],1);
    printf("\n");
	printMat<T,1,grid::NUM_VEL>(&hd_data->h_q_qd_u[grid::NUM_JOINTS+grid::NUM_VEL],1);

	grid::inverse_dynamics_gradient<T,false,false>(hd_data,d_robotModel,gravity,1,dim3(1,1,1),dim3(32,1,1),streams);
	printf("\n");
	printMat<T,grid::NUM_VEL,grid::NUM_VEL>(hd_data->h_dc_du,grid::NUM_VEL);
	printf("\n");
	printMat<T,grid::NUM_VEL,grid::NUM_VEL>(&hd_data->h_dc_du[grid::NUM_VEL*grid::NUM_VEL],grid::NUM_VEL);

	printf("\n");
	grid::direct_minv<T,false>(hd_data,d_robotModel,1,dim3(1,1,1),dimms,streams);
	printMat<T,grid::NUM_VEL,grid::NUM_VEL>(hd_data->h_Minv,grid::NUM_VEL);

	printf("\n");
	grid::forward_dynamics<T>(hd_data,d_robotModel,gravity,1,dim3(1,1,1),dimms,streams);
	printMat<T,1,grid::NUM_VEL>(hd_data->h_qdd,1);

	printf("\n");
	grid::inverse_dynamics<T,false,false>(hd_data,d_robotModel,gravity,1,dim3(1,1,1),dimms,streams);
	printMat<T,1,grid::NUM_VEL>(hd_data->h_c,1);

	grid::forward_dynamics_gradient<T,false>(hd_data,d_robotModel,gravity,1,dim3(1,1,1),dim3(32,1,1),streams);
	printf("\n");
	printMat<T,grid::NUM_VEL,grid::NUM_VEL>(hd_data->h_df_du,grid::NUM_VEL);
	printf("\n");
	printMat<T,grid::NUM_VEL,grid::NUM_VEL>(&hd_data->h_df_du[grid::NUM_VEL*grid::NUM_VEL],grid::NUM_VEL);
	

		// printf("\n");
		// grid::end_effector_positions<T,false>(hd_data,d_robotModel,1,dim3(1,1,1),dimms,streams);
		// printMat<T,1,6*grid::NUM_EES>(hd_data->h_eePos,1);

		// printf("\n");
		// grid::end_effector_positions_gradient<T,false>(hd_data,d_robotModel,1,dim3(1,1,1),dimms,streams);
		// printMat<T,6,grid::NUM_EES*grid::NUM_JOINTS>(hd_data->h_deePos,6);

		// grid::aba<T>(hd_data,d_robotModel,gravity,1,dim3(1,1,1),dimms,streams);
		// printf("\n");
		// printMat<T,1,grid::NUM_JOINTS>(hd_data->h_qdd,1);

		// grid::crba<T>(hd_data,d_robotModel,gravity,1,dim3(1,1,1),dimms,streams);
		// printf("\n");
		// printMat<T,grid::NUM_JOINTS,grid::NUM_JOINTS>(hd_data->h_M,grid::NUM_JOINTS);
	

	grid::close_grid<T>(streams,d_robotModel,hd_data);
}

int main(){
	test<float>(); return 0;
}